#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        const int threadsPerBlock = 256;

        int* d_idata;
        int* d_odata;
        int* d_bools;
        int* d_indices;

        __global__ void kernUpSweep(int n, int depth, int* data) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            int step = 1 << depth;
            int sumOffset = 1 << (depth + 1);

            int i = index * sumOffset + sumOffset - 1;
            data[i] += data[i - step];
        }

        __global__ void kernDownSweep(int n, int depth, int* data) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            int step = 1 << (depth + 1);
            int halfStep = 1 << depth;

            int right = index * step + step - 1;
            int left = index * step + halfStep - 1;

            int tmp = data[left];
            data[left] = data[right];
            data[right] += tmp;
        }

        void scanWithoutTimer(int n, int *data) {
            for (int d = 0; d <= ilog2ceil(n) - 1; ++d) {
                int activeThreads = n >> (d + 1);
                int numBlocks = (activeThreads + threadsPerBlock - 1) / threadsPerBlock;
                kernUpSweep <<<numBlocks, threadsPerBlock>>> (activeThreads, d, data);
            }

            hipMemset(data + n - 1, 0, sizeof(int));
            checkCUDAErrorFn("hipMemset d_odata + newSize - 1 failed.");

            for (int d = ilog2ceil(n) - 1; d >= 0; --d) {
                int activeThreads = n >> (d + 1);
                int numBlocks = (activeThreads + threadsPerBlock - 1) / threadsPerBlock;
                kernDownSweep <<<numBlocks, threadsPerBlock>>> (activeThreads, d, data);
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {
            size_t newSize = 1 << ilog2ceil(n);

            hipMalloc((void**)&d_odata, newSize * sizeof(int));
            checkCUDAErrorFn("hipMalloc d_odata failed.");
            hipMemset(d_odata, 0, newSize * sizeof(int));
            checkCUDAErrorFn("hipMemset d_odata failed.");
            hipMemcpy(d_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("hipMemcpy d_odata <- idata failed.");

            timer().startGpuTimer();

            scanWithoutTimer(newSize, d_odata);

            timer().endGpuTimer();

            hipMemcpy(odata, d_odata, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAErrorFn("hipMemcpy odata <- d_odata failed.");
            hipFree(d_odata);
            checkCUDAErrorFn("hipFree d_odata failed.");
        }
        
        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            size_t newSize = 1 << ilog2ceil(n);

            hipMalloc((void**)&d_idata, newSize * sizeof(int));
            checkCUDAErrorFn("hipMalloc d_idata failed.");
            hipMalloc((void**)&d_odata, newSize * sizeof(int));
            checkCUDAErrorFn("hipMalloc d_odata failed.");
            hipMalloc((void**)&d_bools, newSize * sizeof(int));
            checkCUDAErrorFn("hipMalloc d_bools failed.");
            hipMalloc((void**)&d_indices, newSize * sizeof(int));
            checkCUDAErrorFn("hipMalloc d_indices failed.");

            hipMemset(d_idata, 0, newSize * sizeof(int));
            checkCUDAErrorFn("hipMemset d_idata failed.");
            hipMemcpy(d_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("hipMemcpy d_idata <- idata failed.");

            const int threadsPerBlock = 256;
            int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

            timer().startGpuTimer();
        
            StreamCompaction::Common::kernMapToBoolean <<<numBlocks, threadsPerBlock>>> (newSize, d_bools, d_idata);
            hipMemcpy(d_indices, d_bools, newSize * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAErrorFn("hipMemcpy d_indices <- d_bools failed.");
            scanWithoutTimer(newSize, d_indices);
            StreamCompaction::Common::kernScatter <<<numBlocks, threadsPerBlock>>> (newSize, d_odata, d_idata, d_bools, d_indices);

            timer().endGpuTimer();

            hipMemcpy(odata, d_odata, newSize * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("hipMemcpy odata <- d_odata failed.");

            int count = 0;
            hipMemcpy(&count, d_indices + newSize - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("hipMemcpy count <- d_indices + newSize - 1 failed.");

            hipFree(d_idata);
            checkCUDAErrorFn("hipFree d_idata failed.");
            hipFree(d_odata);
            checkCUDAErrorFn("hipFree d_odata failed.");
            hipFree(d_bools);
            checkCUDAErrorFn("hipFree d_bools failed.");
            hipFree(d_indices);
            checkCUDAErrorFn("hipFree d_indices failed.");

            return count;
        }
    }
}
